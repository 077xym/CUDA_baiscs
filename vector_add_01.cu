#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cmath>

// define the vector size
#define N 10000000
// define the block size (how many threads within a block)
#define BLOCK_SIZE 256

// CPU vector addition
void vector_add_cpu(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for GPU vector addition
__global__ void vector_add_gpu(float *a, float *b, float *c, int n) {
    /*
        grid dim: ceil(N/BLOCK_SIZE) x 1 x 1
        block dim: BLOCK_SIZE x 1 x 1
        
        workflow:
        obtain the offset of each thread, which is the idx to access a and b
        remember to check whether the offset is larger than n or not

        Since whatever the grid and dim size we chose, they will implicitly be transformed to
        3D, we can always use what is shown in 01_index.cu to get offset. However, we reduce
        our operation counts because we know some of the dim will be statically 1.
    */
    int thread_offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_offset < n) {
        c[thread_offset] = a[thread_offset] + b[thread_offset];
    }   
}

// Init vector with random values
void init_vector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

// measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

// a complete view of how to implement vector addition using CUDA
int main() {
    
    // claim the host data variables, remember, these are the data stored in cpu
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
    // claim the device data variables, these are the data stored in gpu, copied from cpu
    float *d_a, *d_b, *d_c;
    // size_t, how many mem are going to be allocated
    size_t size = N * sizeof(float);

    // Allocate host memory (malloc returns void*, so we need to cast its type)
    h_a = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_c_cpu = (float *)malloc(size);
    h_c_gpu = (float *)malloc(size);

    // randomly initialize the vector
    srand(time(NULL)); // set the seed
    init_vector(h_a, N);
    init_vector(h_b, N);

    // Allocate memory on device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // copy data from cpu to gpu
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    /*
        define grid dimension. In this code snippet, we want our grid and block both have 
        1D dimensions. Note, instead of being 1 x BLOCK_SIZE, the actual block dim is implicitly
        BLOCK_SIZE x 1 x 1. So as the grid dim. The number in between <<< >>> tells the nvcc
        the grid and block dim.
    */
    /*
        initially, I wrote int grid_size = ceil(N / BLOCK_SIZE)
        This is wrong, as N and BLOCK_SIZE are both int, N / BLOCK_SIZE has already been floored
        making ceil useless here
    */
    int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    printf("%d\n", grid_size);
    printf("Benchmarking GPU implementation...\n");
    double gpu_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double stime = get_time();
        vector_add_gpu<<<grid_size, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
        hipDeviceSynchronize();
        double etime = get_time();
        gpu_time += etime - stime;
    }
    printf("GPU average time: %f seconds\n", gpu_time / 20.0);

    // execute cpu-version vector addition, and report the time
    printf("Benchmarking CPU implementation...\n");
    double cpu_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double stime = get_time();
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        double etime = get_time();
        cpu_time += etime - stime;
    }
    printf("CPU average time: %f seconds\n", cpu_time / 20.0);

    // check results are equal
    hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu[i]) > 1e-2) {
            correct = false;
            printf("index: %d incorrect. cpu: %f, gpu: %f\n", i, h_c_cpu[i], h_c_gpu[i]);
            break;
        }
    }
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    // free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
